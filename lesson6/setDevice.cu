
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
    // 检测计算机GPU的数量
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount);

    if(error != hipSuccess || iDeviceCount == 0)
    {
        printf("no CUDA compatible GPU device found.\n");
        exit(-1);
    }
    else
    {
        printf("The count of GPUs is %d.\n",iDeviceCount);
    }

    //设置执行的GPU
    int iDev = 0;
    error = hipSetDevice(iDev);
    if(error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing.\n");
    }
    else
    {
        printf("GPU 0 is on ready for computing.\n");
    }

    return 0;
}