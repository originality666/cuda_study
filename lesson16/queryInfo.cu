/*********************************************************************************************
 * file name  : queryInfo.cu
 * author     : 权 双
 * date       : 2023-12-30
 * brief      : GPU指标查询
***********************************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>
#include "common.cuh"


int main(int argc, char **argv)
{ 
    
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProps, devID));
    std::cout << "运行GPU设备:" << deviceProps.name << std::endl;
    std::cout << "SM数量：" << deviceProps.multiProcessorCount << std::endl;
    std::cout << "L2缓存大小：" << deviceProps.l2CacheSize / (1024 * 1024) << "M" << std::endl;
    std::cout << "SM最大驻留线程数量：" << deviceProps.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "设备是否支持流优先级：" << deviceProps.streamPrioritiesSupported << std::endl;
    std::cout << "设备是否支持在L1缓存中缓存全局内存：" << deviceProps.globalL1CacheSupported << std::endl;
    std::cout << "设备是否支持在L1缓存中缓存本地内存：" << deviceProps.localL1CacheSupported << std::endl;
    std::cout << "一个SM可用的最大共享内存量：" << deviceProps.sharedMemPerMultiprocessor / 1024  << "KB" << std::endl;
    std::cout << "一个SM可用的32位最大寄存器数量：" << deviceProps.regsPerMultiprocessor / 1024 << "K" << std::endl;
    std::cout << "一个SM最大驻留线程块数量：" << deviceProps.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "GPU内存带宽：" << deviceProps.memoryBusWidth << std::endl;
    std::cout << "GPU内存频率：" << (float)deviceProps.memoryClockRate / (1024 * 1024) << "GHz" << std::endl;

    

    CUDA_CHECK(hipDeviceReset());

    return 0;
}