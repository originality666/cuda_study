#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include "utils.hpp"

__global__ void ReduceNeighboredWithDivergence(float *d_idata, float *d_odata, int size){
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = d_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= size) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if ((tid % (2 * stride)) == 0)
        {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) d_odata[blockIdx.x] = idata[0];
}

__global__ void ReduceNeighboredWithoutDivergence(float *d_idata, float *d_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = d_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        // convert tid into local array index
        int index = 2 * stride * tid;

        if (index < blockDim.x)
        {
            idata[index] += idata[index + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) d_odata[blockIdx.x] = idata[0];
}


void ReduceOnGPUWithDivergence(float *h_idata, float *h_odata, int size, int blockSize)
{
    int ibytes = size * sizeof(float);
    int obytes = size / blockSize * sizeof(float);

    memset(h_odata, 0, obytes);

    float* d_idata = nullptr;
    float* d_odata = nullptr;

    CUDA_CHECK(hipMalloc(&d_idata, ibytes));
    CUDA_CHECK(hipMalloc(&d_odata, obytes));

    CUDA_CHECK(hipMemcpy(d_idata, h_idata, ibytes, hipMemcpyHostToDevice));
   
    dim3 block(blockSize);
    dim3 grid(size / blockSize);
    ReduceNeighboredWithDivergence <<<grid, block>>> (d_idata, d_odata, size);

    // 将结果从device拷贝回host
    CUDA_CHECK(hipMemcpy(h_odata, d_odata, obytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    //注意在同步后，检测核函数
    CUDA_KERNEL_CHECK();  

    CUDA_CHECK(hipFree(d_odata));
    CUDA_CHECK(hipFree(d_idata));
}

void ReduceOnGPUWithoutDivergence(float *h_idata, float *h_odata, int size, int blockSize)
{
    int ibytes = size * sizeof(float);
    int obytes = size / blockSize * sizeof(float);

    memset(h_odata, 0, obytes);
    
    float* d_idata = nullptr;
    float* d_odata = nullptr;

    CUDA_CHECK(hipMalloc(&d_idata, ibytes));
    CUDA_CHECK(hipMalloc(&d_odata, obytes));

    CUDA_CHECK(hipMemcpy(d_idata, h_idata, ibytes, hipMemcpyHostToDevice));
    
    dim3 block(blockSize);
    dim3 grid(size / blockSize);
    ReduceNeighboredWithoutDivergence <<<grid, block>>> (d_idata, d_odata, size);

    CUDA_CHECK(hipMemcpy(h_odata, d_odata, obytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_KERNEL_CHECK();

    CUDA_CHECK(hipFree(d_odata));
    CUDA_CHECK(hipFree(d_idata));
}

