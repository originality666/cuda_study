
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    const int id  = bid * gridDim.x * gridDim.y * gridDim.z + tid;

    printf("print from block_id %d, thread id %d, global id %d.\n",bid, tid, id);
}

int main(void)
{
    dim3 grid_size(3,3,3);
    dim3 block_size(3,3,3);

    printf("print from cpu!!!");

    hello_from_gpu<<<grid_size, block_size>>>();
    hipDeviceSynchronize();

    return 0;
}