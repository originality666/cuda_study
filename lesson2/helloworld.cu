
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("hello world from gpu.\n");
}

int main(void)
{
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}