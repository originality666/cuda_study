
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    printf("hello world from block_id %d, thread id %d, global id %d.\n",bid, tid, id);
}

int main(void)
{
    hello_from_gpu<<<3, 4>>>();
    hipDeviceSynchronize();

    return 0;
}